#include "hip/hip_runtime.h"
#include <gpu_utils.h>

// =============================================================================
// =============================== MAIN UTILS ==================================
// =============================================================================

static __device__ float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
            __float_as_int(fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void gpuComputeLaserScan(float *d_input_cloud, float *d_transform,
                                    float tolerance, float robot_height,
                                    float *laser_scan)
{
    //Calculate indicies
    int index_x = blockIdx.x*blockDim.x + threadIdx.x;
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;
    int index = index_y*blockDim.x*gridDim.x+ index_x;

    float x = 0.0f;
    float y = 0.0f;
    float z = 0.0f;

    // Shared memory for performance reasons
    __shared__ float sh_TransfMatrix[16];

    // Each thread from specific warp writes to shared matrix
    if(threadIdx.y == 0 && threadIdx.x<16)
        sh_TransfMatrix[threadIdx.x] = d_transform[threadIdx.x];
    __syncthreads();

    // Check if number if not a nan
    if (d_input_cloud[index*4]==d_input_cloud[index*4])
    {

        //rotation P = R x P;
        x = sh_TransfMatrix[0]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[1]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[2]*d_input_cloud[index*4 + 2];

        y = sh_TransfMatrix[4]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[5]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[6]*d_input_cloud[index*4 + 2];

        z = sh_TransfMatrix[8]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[9]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[10]*d_input_cloud[index*4 + 2];

        //translation P = P + t;
        x += sh_TransfMatrix[3];
        y += sh_TransfMatrix[7];
        z += sh_TransfMatrix[11];
    }

    // Check distance
//    if (z<-tolerance || (z>tolerance && z<robot_height))
    if (z>tolerance && z<robot_height)
    {
        int laser_scan_index = (int)((HORIZONTAL_VIEW_RAD/2 + atan2(y, x))/ANGLE_INCREMENT_RAD + 0.5*ANGLE_INCREMENT_RAD);
        atomicMin(&laser_scan[laser_scan_index], sqrt(x*x + y*y));
    }
}

// =============================================================================
// =============================== HELP UTILS ==================================
// =============================================================================

__global__ void gpuExtractCloudFloor(float *d_input_cloud, float *d_transform, float tolerance,
                                     float *d_output_cloud, int *d_floor_points_number)
{
    //Calculate indicies
    int index_x = blockIdx.x*blockDim.x + threadIdx.x;
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;
    int index = index_y*blockDim.x*gridDim.x+ index_x;

    float x = 0.0f;
    float y = 0.0f;
    float z = 0.0f;

    // Shared memory for performance reasons
    __shared__ float sh_TransfMatrix[16];

    // Each thread from specific warp writes to shared matrix
    if(threadIdx.y == 0 && threadIdx.x<16)
        sh_TransfMatrix[threadIdx.x] = d_transform[threadIdx.x];
    __syncthreads();

    // Check if number if not a nan
    if (d_input_cloud[index*4]==d_input_cloud[index*4])
    {

        //rotation P = R x P;
        x = sh_TransfMatrix[0]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[1]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[2]*d_input_cloud[index*4 + 2];

        y = sh_TransfMatrix[4]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[5]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[6]*d_input_cloud[index*4 + 2];

        z = sh_TransfMatrix[8]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[9]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[10]*d_input_cloud[index*4 + 2];

        //translation P = P + t;
        x += sh_TransfMatrix[3];
        y += sh_TransfMatrix[7];
        z += sh_TransfMatrix[11];
    }

    // Check distance
    if (abs(z)<tolerance)
    {
        int where_to_add = atomicAdd(d_floor_points_number,1);
        d_output_cloud[where_to_add*4 + 0] = x;
        d_output_cloud[where_to_add*4 + 1] = y;
        d_output_cloud[where_to_add*4 + 2] = z;
        d_output_cloud[where_to_add*4 + 3] = 0;
    }
}

__global__ void gpuExtractCloudObstacle(float *d_input_cloud, float *d_transform,
                                        float min_height, float max_height,
                                        float *d_output_cloud, int *d_obstacle_points_number)
{
    //Calculate indicies
    int index_x = blockIdx.x*blockDim.x + threadIdx.x;
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;
    int index = index_y*blockDim.x*gridDim.x+ index_x;

    float x = 0.0f;
    float y = 0.0f;
    float z = 0.0f;

    // Shared memory for performance reasons
    __shared__ float sh_TransfMatrix[16];

    // Each thread from specific warp writes to shared matrix
    if(threadIdx.y == 0 && threadIdx.x<16)
        sh_TransfMatrix[threadIdx.x] = d_transform[threadIdx.x];
    __syncthreads();

    // Check if number if not a nan
    if (d_input_cloud[index*4]==d_input_cloud[index*4])
    {

        //rotation P = R x P;
        x = sh_TransfMatrix[0]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[1]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[2]*d_input_cloud[index*4 + 2];

        y = sh_TransfMatrix[4]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[5]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[6]*d_input_cloud[index*4 + 2];

        z = sh_TransfMatrix[8]*d_input_cloud[index*4 + 0] +
            sh_TransfMatrix[9]*d_input_cloud[index*4 + 1] +
            sh_TransfMatrix[10]*d_input_cloud[index*4 + 2];

        //translation P = P + t;
        x += sh_TransfMatrix[3];
        y += sh_TransfMatrix[7];
        z += sh_TransfMatrix[11];
    }

    // Check distance
    if (z>min_height && z<max_height)
    {
        int where_to_add = atomicAdd(d_obstacle_points_number,1);
        d_output_cloud[where_to_add*4 + 0] = x;
        d_output_cloud[where_to_add*4 + 1] = y;
        d_output_cloud[where_to_add*4 + 2] = z;
        d_output_cloud[where_to_add*4 + 3] = 0;
    }
}

// =============================================================================
// =============================== CON & DES ===================================
// =============================================================================

GPUUtils::GPUUtils(bool sensor_1_, bool sensor_2_)
{
    // Alocate gpu memory
    sensor_1 = sensor_1_;
    sensor_2 = sensor_2_;
    if (sensor_1)
    {
        hipMalloc((void**)&d_input_cloud_1, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float));
        hipMalloc((void**)&d_transformation_1, 16*sizeof(float));
    }
    if (sensor_2)
    {
        hipMalloc((void**)&d_input_cloud_2, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float));
        hipMalloc((void**)&d_transformation_2, 16*sizeof(float));
    }
    hipMalloc((void**)&d_laser_scan, LASER_SCANNER_POINTS*sizeof(float));

    // Kernel invocation params
    dim_block = dim3(32,32,1);                             // 32x32 = 1024
    dim_grid  = dim3(CLOUD_WIDTH/32,CLOUD_HEIGHT/32,1);    // 620/32 = 20, 480/32=15
}
GPUUtils::~GPUUtils()
{
    // Dealocate gpu memory
    if (sensor_1)
    {
        hipFree((void*)d_input_cloud_1);
        hipFree((void*)d_transformation_1);
    }
    if (sensor_2)
    {
        hipFree((void*)d_input_cloud_2);
        hipFree((void*)d_transformation_2);
    }
    hipFree((void*)d_laser_scan);
}

// =============================================================================
// ============================== LASER SCAN ===================================
// =============================================================================

void GPUUtils::computeLaserScan(const unsigned char *input_cloud, float *transformation,
                                float tolerance, float robot_height, float* laser_scan)
{
    // Copy memory to GPU
    hipMemcpy((void*)d_input_cloud_1, (void*)input_cloud, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_transformation_1, transformation, 16*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_laser_scan, (void*)laser_scan, LASER_SCANNER_POINTS*sizeof(float), hipMemcpyHostToDevice);

    // Kernel
    gpuComputeLaserScan<<<dim_grid,dim_block>>>(d_input_cloud_1, d_transformation_1,
                                                tolerance, robot_height,
                                                d_laser_scan);

    // Copy memory back to CPU
    hipMemcpy((void*)laser_scan, (void*)d_laser_scan, LASER_SCANNER_POINTS*sizeof(float), hipMemcpyDeviceToHost);
}

void GPUUtils::computeLaserScan(const unsigned char *input_cloud_1, float *transformation_1,
                                const unsigned char *input_cloud_2, float *transformation_2,
                                float tolerance, float robot_height, float* laser_scan)
{
    // Time measurement
//    float a,b,c;
//    hipEvent_t time_start, time_memcpyH2D, time_kernel, time_memcpyD2H;
//    hipEventCreate(&time_start);
//    hipEventCreate(&time_memcpyH2D);
//    hipEventCreate(&time_kernel);
//    hipEventCreate(&time_memcpyD2H);
//    hipEventRecord(time_start,0);

    // Copu memory to GPU
    hipMemcpy((void*)d_input_cloud_1, (void*)input_cloud_1, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_input_cloud_2, (void*)input_cloud_2, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_transformation_1, transformation_1, 16*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_transformation_2, transformation_2, 16*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_laser_scan, (void*)laser_scan, LASER_SCANNER_POINTS*sizeof(float), hipMemcpyHostToDevice);
//    hipEventRecord(time_memcpyH2D,0);

    // Kernel
    gpuComputeLaserScan<<<dim_grid,dim_block>>>(d_input_cloud_1, d_transformation_1,
                                                tolerance, robot_height,
                                                d_laser_scan);
    gpuComputeLaserScan<<<dim_grid,dim_block>>>(d_input_cloud_2, d_transformation_2,
                                                tolerance, robot_height,
                                                d_laser_scan);
//    hipEventRecord(time_kernel,0);

    // Copy back and free
    hipMemcpy((void*)laser_scan, (void*)d_laser_scan, LASER_SCANNER_POINTS*sizeof(float), hipMemcpyDeviceToHost);

    //Time
//    hipEventRecord(time_memcpyD2H,0);
//    hipEventSynchronize(time_memcpyD2H);
//    hipEventElapsedTime(&a, time_start,time_memcpyH2D);
//    hipEventElapsedTime(&b, time_memcpyH2D,time_kernel);
//    hipEventElapsedTime(&c, time_kernel,time_memcpyD2H);

//    printf("MemoryH2D                 =%f\n", a);
//    printf("Kernel                    =%f\n", b);
//    printf("MemoryD2H                 =%f\n", c);
}

// =============================================================================
// ================================= DEBUG =====================================
// =============================================================================

void GPUUtils::extractCloudFloor(const unsigned char *input_cloud, float *transformation, float floor_tolerance,
                                 unsigned char **output_cloud, int &cloud_floor_points)
{
    // Alocate gpu memory and copy
    float *d_output_cloud;
    int *d_floor_points_number;
    hipMalloc((void**)&d_output_cloud, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float));
    hipMalloc((void**)&d_floor_points_number, sizeof(int));
    hipMemcpy((void*)d_input_cloud_1, (void*)input_cloud, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_transformation_1, transformation, 16*sizeof(float), hipMemcpyHostToDevice);

    // Pre-initialize memory for grid cells
    hipMemset((void*)d_floor_points_number, 0, sizeof(int));
    hipMemset((void*)d_output_cloud, 0, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(int));

    // GPU kernel
    gpuExtractCloudFloor<<<dim_grid,dim_block>>>(d_input_cloud_1, d_transformation_1,
                                                 floor_tolerance, d_output_cloud,
                                                 d_floor_points_number);

    // Copy interesing data
    hipMemcpy((void*)&cloud_floor_points,(void*)d_floor_points_number,sizeof(int),hipMemcpyDeviceToHost);
    *output_cloud = new unsigned char[16*cloud_floor_points];
    hipMemcpy((void*)*output_cloud, (void*)d_output_cloud, 4*cloud_floor_points*sizeof(float), hipMemcpyDeviceToHost);

    // Free the gpu resources
    hipFree((void*)d_output_cloud);
    hipFree((void*)d_floor_points_number);
}

void GPUUtils::extractCloudObstacle(const unsigned char *input_cloud, float *transformation, float min_height, float max_height,
                                   unsigned char **output_cloud, int &cloud_obstacle_points)
{
    // Alocate gpu memory and copy
    float *d_output_cloud;
    int *d_obstacle_points_number;
    hipMalloc((void**)&d_output_cloud, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float));
    hipMalloc((void**)&d_obstacle_points_number, sizeof(int));
    hipMemcpy((void*)d_input_cloud_1, (void*)input_cloud, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_transformation_1, transformation, 16*sizeof(float), hipMemcpyHostToDevice);

    // Pre-initialize memory for grid cells
    hipMemset((void*)d_obstacle_points_number, 0, sizeof(int));
    hipMemset((void*)d_output_cloud, 0, CLOUD_WIDTH*CLOUD_HEIGHT*4*sizeof(int));

    // GPU kernel
    gpuExtractCloudObstacle<<<dim_grid,dim_block>>>(d_input_cloud_1, d_transformation_1,
                                                    min_height, max_height, d_output_cloud,
                                                    d_obstacle_points_number);

    // Copy interesing data
    hipMemcpy((void*)&cloud_obstacle_points, (void*)d_obstacle_points_number, sizeof(int),hipMemcpyDeviceToHost);
    *output_cloud = new unsigned char[16*cloud_obstacle_points];
    hipMemcpy((void*)*output_cloud, (void*)d_output_cloud, 4*cloud_obstacle_points*sizeof(float), hipMemcpyDeviceToHost);

    // Free the gpu resources
    hipFree((void*)d_output_cloud);
    hipFree((void*)d_obstacle_points_number);
}

